#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../../commons/h5bench_util.h"
#include "cuda_kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void kernel(float *d_x, float *d_y, float *d_z, float *d_px, float *d_py, float *d_pz,
  int *d_id_1, float *d_id_2, long particle_cnt, volatile int *kernel_flag) {

  int32_t total_threads = blockDim.x * gridDim.x;
  int32_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  for( int32_t i = gid; i < particle_cnt; i+=total_threads ) {
    d_x[i] += 0.2;
    d_y[i] += 0.2;
    d_z[i] += 0.2;
    d_px[i] += 0.2;
    d_py[i] += 0.2;
    d_pz[i] += 0.2;
    d_id_1[i] += 1;
    d_id_2[i] += 0.2;
  }

  // wait for cpu to tell kernel to finish, or keep running
  while(!*kernel_flag);
}

void kernel_call(data_contig_md *data, volatile int *kernel_flag, hipStream_t stream_id) {
  dim3 threadsperblock = 128;
  dim3 blockspergrid = 80;

  //dim3 blockspergrid = dim3(ceil((double)numparticles/256), 1, 1);
  //printf("cuda kernel launch with %d blocks of %d threads\n", blockspergrid, threadsperblock);
  // kernel<<<threadsperblock, blockspergrid, 0, stream_id>>>(data, kernel_flag);

  // kernel<<<threadsperblock, blockspergrid, 0, stream_id>>>(
    // data->d_x, data->d_y, data->d_z, data->d_px, data->d_py, data->d_pz, data->d_id_1, data->d_id_2, data->particle_cnt,
    // kernel_flag);

  // todo: false postive cufile error?
  //runtime_api_call(cudapeekatlasterror());
}
